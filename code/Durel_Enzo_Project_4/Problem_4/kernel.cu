
#include <hip/hip_runtime.h>
__global__
void blur_kernel(int* in, int* out, int* filter, int w, int h, int filter_size)
{
    int Col = blockIdx.x * blockDim.x + threadIdx.x;
    int Row = blockIdx.y * blockDim.y + threadIdx.y;
    int blurSize = filter_size / 2;

    if (Col < w && Row < h)
    {
	int pixVal = 0;
	
	for (int blurRow = -blurSize; blurRow < blurSize+1; ++blurRow)
	{
	    for (int blurCol = -blurSize; blurCol < blurSize+1; ++blurCol)
	    {
		int curRow = Row + blurRow;
		int curCol = Col + blurCol;

		if (curRow > -1 && curRow < h && curCol > -1 && curCol < w)
		{
		    int filter_x = blurCol + blurSize;
		    int filter_y = blurRow + blurSize;
		    int filter_value = filter[filter_y * filter_size + filter_x];
		    pixVal += in[curRow * w + curCol] * filter_value;
		}
	    }
	}

	out[Row * w + Col] = pixVal;
    }
}

__global__
void maxpooling_kernel(int* in, int* out, int w, int h, int pool_size)
{
    int Col = blockIdx.x * blockDim.x + threadIdx.x;
    int Row = blockIdx.y * blockDim.y + threadIdx.y;
    int half = pool_size / 2;

    if (Col < w && Row < h)
    {
        int maxVal = INT_MIN;

        for (int dy = -half; dy <= half; ++dy)
        {
            for (int dx = -half; dx <= half; ++dx)
            {
                int curRow = Row + dy;
                int curCol = Col + dx;

                if (curRow >= 0 && curRow < h && curCol >= 0 && curCol < w)
                {
                    int val = in[curRow * w + curCol];
                    if (val > maxVal)
                        maxVal = val;
                }
            }
        }

        out[Row * w + Col] = maxVal;
    }
}
