#include "hip/hip_runtime.h"
__global__
void blur_kernel(int* in, int* out, int* filter, int w, int h, int filter_size)
{
    int Col = blockIdx.x * blockDim.x + threadIdx.x;
    int Row = blockIdx.y * blockDim.y + threadIdx.y;
    int blurSize = filter_size / 2;

    if (Col < w && Row < h)
    {
	int pixVal = 0;
	
	for (int blurRow = -blurSize; blurRow < blurSize+1; ++blurRow)
	{
	    for (int blurCol = -blurSize; blurCol < blurSize+1; ++blurCol)
	    {
		int curRow = Row + blurRow;
		int curCol = Col + blurCol;

		if (curRow > -1 && curRow < h && curCol > -1 && curCol < w)
		{
		    int filter_x = blurCol + blurSize;
		    int filter_y = blurRow + blurSize;
		    int filter_value = filter[filter_y * filter_size + filter_x];
		    pixVal += in[curRow * w + curCol] * filter_value;
		}
	    }
	}

	out[Row * w + Col] = pixVal;
    }
}

__global__
void maxpooling_kernel(int* in, int* out, int w, int h, int maxpooling_size)
{
    int Col = blockIdx.x * blockDim.x + threadIdx.x;
    int Row = blockIdx.y * blockDim.y + threadIdx.y;
    int blurSize = filter_size / 2;

    if (Col < w && Row < h)
    {
	int pixVal = 0;
	
	for (int blurRow = -blurSize; blurRow < blurSize+1; ++blurRow)
	{
	    for (int blurCol = -blurSize; blurCol < blurSize+1; ++blurCol)
	    {
		int curRow = Row + blurRow;
		int curCol = Col + blurCol;

		if (curRow > -1 && curRow < h && curCol > -1 && curCol < w)
		{
		    if (outputMatrix_h[curRow*h + curCol] > max)
			max = outputMatrix_h[curRow*h + curCol];
		}
	    }
	}

	out[Row * w + Col] = max;
    }
}
