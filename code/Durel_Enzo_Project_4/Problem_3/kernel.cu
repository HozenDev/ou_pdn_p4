
#include <hip/hip_runtime.h>
__global__
void blur_kernel(int* in, int* out, int* filter, int w, int h, int filter_size)
{
    int Col = blockIdx.x * blockDim.x + threadIdx.x;
    int Row = blockIdx.y * blockDim.y + threadIdx.y;
    int blurSize = filter_size / 2;

    if (Col < w && Row < h)
    {
	int pixVal = 0;
	
	for (int blurRow = -blurSize; blurRow < blurSize+1; ++blurRow)
	{
	    for (int blurCol = -blurSize; blurCol < blurSize+1; ++blurCol)
	    {
		int curRow = Row + blurRow;
		int curCol = Col + blurCol;

		if (curRow > -1 && curRow < h && curCol > -1 && curCol < w)
		{
		    int filter_x = blurCol + blurSize;
		    int filter_y = blurRow + blurSize;
		    int filter_value = filter[filter_y * filter_size + filter_x];
		    pixVal += in[curRow * w + curCol] * filter_value;
		}
	    }
	}

	out[Row * w + Col] = pixVal;
    }
}
