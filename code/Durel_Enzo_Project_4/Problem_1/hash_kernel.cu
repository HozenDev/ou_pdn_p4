
#include <hip/hip_runtime.h>
__device__
unsigned int generate_hash(unsigned int nonce, unsigned int index, unsigned int* transactions, unsigned int n_transactions, unsigned int max)
{
    unsigned int hash = (nonce + transactions[0] * (index + 1)) % max;
    for (int j = 1; j < n_transactions; j++) {
	hash = (hash + transactions[j] * (index + 1)) % max;
    }
    return hash;
}

/* Hash Kernel --------------------------------------
*       Generates an array of hash values from nonces.
*/
__global__
void hash_kernel(unsigned int* hash_array, unsigned int* nonce_array, unsigned int array_size, unsigned int* transactions, unsigned int n_transactions, unsigned int mod) {

    // Calculate thread index
    unsigned int index = blockDim.x * blockIdx.x + threadIdx.x;

    // Calculate hash value
    if (index < array_size) {
	hash_array[index] = generate_hash(nonce_array[index], index, transactions, n_transactions, mod);
    }

} // End Hash Kernel //
